#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__device__ double computeHa(double a){
	double ha;
	if (a < ha_a[0])
		ha = ha_y[0];
	else if (a > ha_a[ha_a_len - 1])
		ha = ha_y[ha_y_len - 1];
	else {
		int l, r, m;
		for (l = 0, r = ha_a_len - 1, m = (l + r) / 2; 1 < r - l; m = (l + r) / 2)
			if (a < ha_a[m])
				r = m;
			else
				l = m;
        ha = (a - ha_a[l]) / (ha_a[r] - ha_a[l]) * ha_y[l] + (ha_a[r] - a) / (ha_a[r] - ha_a[l]) * ha_y[r];
	}
	return ha;
}

__device__ double computeXe(double a){
	double xe;
	if (a < rec_a[0])
		xe = rec_xe[0];
	else if (a > rec_a[rec_a_len - 1])
		xe = rec_xe[rec_xe_len - 1];
	else {
		int l, r, m;
		for (l = 0, r = rec_a_len - 1, m = (l + r) / 2; 1 < r - l; m = (l + r) / 2)
			if (a < rec_a[m])
				r = m;
			else
				l = m;
        xe = (a - rec_a[l]) / (rec_a[r] - rec_a[l]) * rec_xe[l] + (rec_a[r] - a) / (rec_a[r] - rec_a[l]) * rec_xe[r];
	}
	return xe;
}

__device__ double computeEta(double a){
	double eta;
	if (a < ha_a[0])
		eta = eta_a[0];
	else if (a > ha_a[ha_a_len - 1])
		eta = eta_a[eta_a_len - 1];
	else {
		int l, r, m;
		for (l = 0, r = ha_a_len - 1, m = (l + r) / 2; 1 < r - l; m = (l + r) / 2)
			if (a < ha_a[m])
				r = m;
			else
				l = m;
		eta = (a - ha_a[l]) / (ha_a[r] - ha_a[l]) * eta_a[l] + (ha_a[r] - a) / (ha_a[r] - ha_a[l]) * eta_a[r];
	}
	return eta;
}



struct myFex{
    __device__ void operator()(int *neq, double *ts, double *y, double *dy/*, void *otherData*/)
    {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        
        int len = neq[0];
        
        double t = ts[0];
        
        double a = exp(t);
        float k = tex2D(args_tex,1, tid);
        
        double ha = computeHa(a);

        // (see Dodelson eq. 3.45, p73); [h Mpc^-1; caution: a factor of h may be missing; also this neglect He and so needs to be revised
    	double xe = computeXe(a);
    	
        double tdot = -0.0692 / pow(a, 2) * xe * omega_b * h / rh;

		if (len == 7) {
			double dphi = -(1. + pow(k, 2) / (3. * pow(a, 2) * pow(ha, 2))) * y[0]
						+ 0.5 / pow(ha * rh, 2) * (omega_dm * pow(a, -3) * y[1] + omega_b * pow(a, -3) * y[3] + 4. * omega_r * pow(a, -4) * y[5]);
			double r_bph = 3. / 4. * omega_b / omega_gam * a;
			
			dy[0] = dphi; // phi, 0
			dy[1] = -k / (a * ha) * y[2] - 3. * dphi; // delta, 1
			dy[2] = -y[2] - k / (a * ha) * y[0]; // u = i*v, 2
			dy[3] = -k / (a * ha) * y[4] - 3. * dphi; // delta_b, 3
			dy[4] = -y[4] - k / (a * ha) * y[0] + tdot / (r_bph * a * ha) * (y[4] - 3. * y[6]); // u_b, 4
			dy[5] = -k / (a * ha) * y[6] - dphi; // theta0, 5
			dy[6] = k / (3. * a * ha) * (y[5] - y[0]) + tdot / (a * ha) * (y[6] - y[4] / 3.); // theta1, 6
	

		} else {

			int lmax = (len - 5) / 2;
			double eta = computeEta(a);

			int l;

			double psi = -y[0] - 12. * pow(k * a * rh, 2) * omega_r * y[7];
			double dphi = psi - 1./3. * pow(k / (a * ha), 2) * y[0]
						+ 0.5 / pow(ha * rh, 2) * (omega_dm * pow(a, -3) * y[1] + omega_b * pow(a, -3) * y[3] + 4. * omega_r * pow(a, -4) * y[5]);
			double r_bph = 3. / 4. * omega_b / omega_gam * a;
			double Pi = y[7] + y[5 + lmax] + y[5 + lmax + 2];

			dy[0] = dphi; // phi, 0
			dy[1] = -k / (a * ha) * y[2] - 3. * dphi; // delta, 1
			dy[2] = -y[2] + k / (a * ha) * psi; // u = i*v, 2
			dy[3] = -k / (a * ha) * y[4] - 3. * dphi; // delta_b, 3
			dy[4] = -y[4] + k / (a * ha) * psi + tdot / (r_bph * a * ha) * (y[4] - 3. * y[6]); // u_b, 4

			dy[5] = -k / (a * ha) * y[6] - dphi; // theta_0, 5
			dy[6] = 1./3. * k / (a * ha) * (y[5] - y[7] + psi) + tdot / (a * ha) * (y[6] - y[4] / 3.); // theta_1, 6

			dy[7] = 1./5. * k / (a * ha) * (2. * y[6] - 3. * y[8]) + tdot / (a * ha) * (y[7] - Pi / 10.); // theta_2, 7

			l = 0;
			dy[5 + lmax] = k / (a * ha) * 1 / (2 * l + 1) * (-(l + 1)* y[5 + lmax + 1])
					+ tdot / (a * ha) * (y[5 + lmax] - Pi / 2. * 1. ); // theta_p0, 5 + l_max + 0

			l = 1;
			dy[5 + lmax + l] = k / (a * ha) * 1 / (2 * l + 1) * (l * y[5 + lmax + l - 1] - (l + 1) * y[5 + lmax + l + 1])
				+ tdot / (a * ha) * (y[5 + lmax + l]); // theta_p1, 5 + l_max + 1

			l = 2;
			dy[5 + lmax + l] = k / (a * ha) * 1 / (2 * l + 1) * (l * y[5 + lmax + l - 1] - (l + 1) * y[5 + lmax + l + 1])
					+ tdot / (a * ha) * (y[5 + lmax + l] - Pi / 2. * 1. / 5. ); // theta_p2, 5 + l_max + 2

			for (l = 3; l < lmax-1; l++) {
				dy[5 + l] = k / (a * ha) * 1 / (2 * l + 1) * (l * y[5 + l - 1] - (l + 1) * y[5 + l + 1])
					+ tdot / (a * ha) * y[5 + l]; // theta_l, 5 + l

				dy[5 + lmax + l] = k / (a * ha) * 1 / (2 * l + 1) * (l * y[5 + lmax + l - 1] - (l + 1) * y[5 + lmax + l + 1])
					+ tdot / (a * ha) * (y[5 + lmax + l]); // theta_pl, 5 + l_max + l
			}

			l = lmax - 1;

			dy[5 + l] = 1 / (a * ha) * (k * y[5 + l - 1] - ((l * 1) / eta - tdot) * y[5 + l]); // theta_l_max, 5 + lmax

			dy[5 + lmax + l] = 1 / (a * ha) * (k * y[5 + lmax + l - 1] - ((l * 1) / eta - tdot) * y[5 + lmax + l]); // theta_pl_max, 5 + 2*l_max

		}
    }
};

struct myJex{
    __device__ void operator()(int *neq, double *ts, double *y, int ml, int mu, double *dy, int nrowpd/*, void *otherData*/){
    	int tid = blockDim.x * blockIdx.x + threadIdx.x;
    	
        int len = neq[0];
        int lmax = (len - 5) / 2;
        
        double t = ts[0];
        
        double a = exp(t);
        float k = tex2D(args_tex,1, tid);
        
    	double ha = computeHa(a);
        // (see Dodelson eq. 3.45, p73); [h Mpc^-1; caution: a factor of h may be missing; also this neglect He and so needs to be revised
    	double xe = computeXe(a);
		
		double tdot = -0.0692 / pow(a, 2) * xe * omega_b * h / rh;
        double r_bph = 3. / 4. * omega_b / omega_gam * a;
        
        double harh2 = pow(ha * rh, 2);
        double kaha = k / (a * ha);
        double tdotaha = tdot / (a * ha);
        double tdotaha3 = tdotaha / 3;
        double tdotaha10 = tdotaha / 10;
        double a2 = pow(a, 2);
        double a3 = pow(a, 3);
        double am3 = pow(a, -3);
        double am4 = pow(a, -4);
        double ha2 = pow(ha, 2);
        double k2 = pow(k, 2);
        
        double a2k2 = a2 * k2;
        
        double rhi2 = pow(1 / rh, 2);
        
        double harh2_omega_dm_a3 = harh2 * omega_dm * (2 * am3);
        double harh2_omega_b_a3 = harh2 * omega_b * (2 * am3);
        double harh2_omega_r_a4 = harh2 * omega_r * am4;
        

        memset(dy, 0, len*len*sizeof(double));

    	#define I(j, i) (i * len + j)

        int i = 0;
    	dy[I(i, 0)] = -( k2 / (3. * a2 * ha2)) - 1.;
    	dy[I(i, 1)] = 1. / harh2_omega_dm_a3;
    	//dy[I(i, 2)] = 0;
    	dy[I(i, 3)] = 1. / harh2_omega_b_a3;
    	//dy[I(i, 4)] = 0;
    	dy[I(i, 5)] = 2. / harh2_omega_r_a4;
    	//dy[I(i, 6)] = 0;

    	i = 1;
    	dy[I(i, 0)] = ( k2 / (a2 * ha2)) + 3.;
    	dy[I(i, 1)] = -3. / harh2_omega_dm_a3;
    	dy[I(i, 2)] = -kaha;
    	dy[I(i, 3)] = -3. / harh2_omega_b_a3;
    	//dy[I(i, 4)] = 0;
    	dy[I(i, 5)] = -6. / harh2_omega_r_a4;
    	//dy[I(i, 6)] = 0;

    	i = 2;
    	dy[I(i, 0)] = -kaha;
    	//dy[I(i, 1)] = 0;
    	dy[I(i, 2)] = -1;
    	//dy[I(i, 3)] = 0;
    	//dy[I(i, 4)] = 0;
    	//dy[I(i, 5)] = 0;
    	//dy[I(i, 6)] = 0;

    	i = 3;
    	dy[I(i, 0)] = ( k2 / (a2 * ha2)) + 3;
    	dy[I(i, 1)] = -3. / harh2_omega_dm_a3;
    	//dy[I(i, 2)] = 0;
    	dy[I(i, 3)] = -3. / harh2_omega_b_a3;
    	dy[I(i, 4)] = -kaha;
    	dy[I(i, 5)] = -6. / harh2_omega_r_a4;
    	//dy[I(i, 6)] = 0;

    	i = 4;
    	dy[I(i, 0)] = -kaha;
    	//dy[I(i, 1)] = 0;
    	//dy[I(i, 2)] = 0;
    	//dy[I(i, 3)] = 0;
    	dy[I(i, 4)] = -1. + tdot / (r_bph * a * ha);
    	//dy[I(i, 5)] = 0;
    	dy[I(i, 6)] = -3. * tdot / (r_bph * a * ha);

    	i = 5; //dtheta 0
    	dy[I(i, 0)] = ( k2 / (3. * a2 * ha2)) + 1.;
    	dy[I(i, 1)] = -1. / harh2_omega_dm_a3;
    	//dy[I(i, 2)] = 0;
    	dy[I(i, 3)] = -1. / harh2_omega_b_a3;
    	//dy[I(i, 4)] = 0;
    	dy[I(i, 5)] = -2. / harh2_omega_r_a4;
    	dy[I(i, 6)] = -kaha;

    	i = 6; //dtheta 1
    	dy[I(i, 0)] = -k/ (3. * a * ha);
    	//dy[I(i, 1)] = 0;
    	//dy[I(i, 2)] = 0;
    	//dy[I(i, 3)] = 0;
    	dy[I(i, 4)] = -tdotaha3;
    	dy[I(i, 5)] = k / (3. * a * ha);
    	dy[I(i, 6)] = tdotaha;

    	if(len>7){

    		//theta2
    		dy[I(0, 7)] = -12 * rhi2 * omega_r / (a2k2);
    		dy[I(1, 7)] = 36 * rhi2 * omega_r / (a2k2);
    		dy[I(2, 7)] = -12 / harh2 * omega_r / (a3 * ha * k);
    		dy[I(3, 7)] = 36 * rhi2 * omega_r / (a2k2);
    		dy[I(4, 7)] = -12 / harh2 * omega_r / (a3 * ha * k);
    		dy[I(5, 7)] = 12 * rhi2 * omega_r / (a2k2);
    		dy[I(6, 7)] = -(a2k2 + 12 * rhi2 * omega_r) / (3 * a3 * ha * k);

    		if(lmax>3){
    			i = 7; //dtheta 2
//    			dy[I(i, 0)] = 0;
//    			dy[I(7, 1)] = 0;
//    			dy[I(7, 2)] = 0;
//    			dy[I(7, 3)] = 0;
//    			dy[I(7, 4)] = 0;
//    			dy[I(7, 5)] = 0;
    			dy[I(7, 6)] = 2 * k / (5 * a * ha);
    			dy[I(7, 7)] = 9 * tdotaha10;
    			dy[I(7, 8)] = -3 * k / (5 * a * ha);
    			dy[I(7, (5 + lmax + 0))] = -tdotaha10;
    			dy[I(7, (5 + lmax + 2))] = -tdotaha10;
    		}

    		int l;
    		l=0; //dtheta p 0
    		dy[I((5 + lmax + l), 7)] = -tdot / (2 * a * ha);
    		dy[I((5 + lmax + l), (5 + lmax + 0))] = tdot / (2 * a * ha);
    		//dy[I((5 + lmax + l), (5 + lmax + 1))] = (k + k * l) / (a * ha * (2 * l + 1));
    		dy[I((5 + lmax + l), (5 + lmax + 1))] = kaha * l / (1 + 2 * l);
    		dy[I((5 + lmax + l), (5 + lmax + 2))] = -tdot / (2 * a * ha);

    		l=1; //dtheta p 1
    		//dy[I((5 + lmax + l), (5 + lmax + 0))] = (k * l) / (a * ha * (2 * l + 1));
    		dy[I((5 + lmax + l), (5 + lmax + 0))] = kaha * l / (1 + 2 * l);
    		dy[I((5 + lmax + l), (5 + lmax + 1))] = tdotaha;
    		//dy[I((5 + lmax + l), (5 + lmax + 2))] = -(k + k * l) / (a * ha * (2 * l + 1));
    		dy[I((5 + lmax + l), (5 + lmax + 2))] = -kaha * (1 + l) / (1 + 2 * l);

    		if(lmax>3){
    			l=2; //dtheta p 2
    			dy[I((5 + lmax + l), 7)] = -tdotaha10;
    			dy[I((5 + lmax + l), (5 + lmax + 0))] = -tdotaha10;
    			//dy[I((5 + lmax + l), (5 + lmax + 1))] = (k * l) / (a * ha * (2 * l + 1));
    			dy[I((5 + lmax + l), (5 + lmax + 1))] = kaha * l / (1 + 2 * l);
    			dy[I((5 + lmax + l), (5 + lmax + 2))] = 9 * tdotaha10;
    			//dy[I((5 + lmax + l), (5 + lmax + 3))] = -(k + k * l) / (a * ha * (2 * l + 1));
    			dy[I((5 + lmax + l), (5 + lmax + 3))] = -kaha * (1 + l) / (1 + 2 * l);
    		}

    		double dthetalm1 = 0;
    		double dthetalp1 = 0;
    		for (l = 3; l < lmax-1; l++) {
    			//dtheta L
    			//dy[I((5 + l), (5 + l - 1))] = (k * l) / (a * ha * (2 * l + 1));
    			dthetalm1 = kaha * l / (1 + 2 * l);
    			dy[I((5 + l), (5 + l - 1))] = dthetalm1;
    			dy[I((5 + l), (5 + l))] = tdotaha;
    			//dy[I((5 + l), (5 + l + 1))] = -(k + k * l) / (a * ha * (2 * l + 1));
    			dthetalp1 = -kaha * (1 + l) / (1 + 2 * l);
    			dy[I((5 + l), (5 + l + 1))] = dthetalp1;

    			//dtheta P L
    			//dy[I((5 + lmax + l), (5 + lmax + l - 1))] = (k * l) / (a * ha * (2 * l + 1));
    			dy[I((5 + lmax + l), (5 + lmax + l - 1))] = dthetalm1;
    			dy[I((5 + lmax + l), (5 + lmax + l))] = tdotaha;
    			//dy[I((5 + lmax + l), (5 + lmax + l + 1))] = -(k + k * l) / (a * ha * (2 * l + 1));
    			dy[I((5 + lmax + l), (5 + lmax + l + 1))] = dthetalp1;
    		}


    		double eta = computeEta(a);
    		
    		l = lmax - 1;
    		//dtheta lmax
    		//reset previously set values in this row
    		memset(&dy[I((5 + l), 0)], 0, len*sizeof(double));
    		dy[I((5 + l), (5 + l - 1))] = kaha;
    		dy[I((5 + l), (5 + l))] = 1 / (a * ha) * (-(l + 1) / eta + tdot);

    		//dtheta p lmax
    		//reset previously set values in this row
    		memset(&dy[I((5 + lmax + l), 0)], 0, len*sizeof(double));
    		dy[I((5 + lmax + l), (5 + lmax + l - 1))] = kaha;
    		dy[I((5 + lmax + l), (5 + lmax + l))] = 1 / (a * ha) * (-(l + 1) / eta + tdot);

    	}

    	#undef I
        return; 
    }
};