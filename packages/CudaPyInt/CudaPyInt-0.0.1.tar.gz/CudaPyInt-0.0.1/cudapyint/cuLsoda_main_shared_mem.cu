#include "hip/hip_runtime.h"
    extern "C"{
    
        __device__ myFex myfex;
        __device__ myJex myjex;
        
        __global__ void init_common(){
            int tid = blockDim.x * blockIdx.x + threadIdx.x;
            cuLsodaCommonBlockInit( &(common[tid]) );
        }
        
        __global__ void cuLsoda(const int *g_neq, double *g_y, double *g_t,
        		double *g_tout, const int *g_itol, const double *g_rtol, double *atol, const int *g_itask,
        		int *g_istate, const int *g_iopt, double *rwork, const int *g_lrw, int *iwork,
        		const int *g_liw, const int *g_jt, int *isize, int *rsize) {

        	int tid = blockDim.x * blockIdx.x + threadIdx.x;
        	__shared__ int neq;
        	neq = g_neq[0];
        	__shared__ int itol;
        	itol = g_itol[0];
        	__shared__ double rtol;
        	rtol = g_rtol[0];
        	__shared__ int itask;
        	itask = g_itask[0];
        	__shared__ int iopt;
        	iopt = g_iopt[0];
        	__shared__ int lrw;
        	lrw = g_lrw[0];
        	__shared__ int liw;
        	liw = g_liw[0];
        	__shared__ int jt;
        	jt = g_jt[0];

        	__shared__ double t[BLOCK_SIZE];
        	t[threadIdx.x] = g_t[tid];
        	__shared__ double tout[BLOCK_SIZE];
        	tout[threadIdx.x] = g_tout[tid];
        	__shared__ int istate[BLOCK_SIZE];
        	istate[threadIdx.x] = g_istate[tid];

        	__shared__ double y[BLOCK_SIZE*NEQ];
        	memcpy(y+threadIdx.x*neq, g_y+tid*neq, sizeof(double)*neq);


        	//printf("*tid:%i t:%4.2f t:%4.2f \t y=[%G %G %G %G %G %G %G]\n", tid, t[tid], tout[tid], y[threadIdx.x*neq+0], y[threadIdx.x*neq+1], y[threadIdx.x*neq+2], y[threadIdx.x*neq+3], y[threadIdx.x*neq+4], y[threadIdx.x*neq+5], y[threadIdx.x*neq+6]);
        	//printf("*tid:%i t:%4.2f %4.2f t:%4.2f %4.2f \n", tid, t[tid], g_t[tid], tout[tid], g_tout[tid]);

        	dlsoda_(myfex, &neq, &y[threadIdx.x*neq], &t[threadIdx.x], &tout[threadIdx.x], &itol, &rtol,
        			atol, &itask, &istate[threadIdx.x], &iopt, rwork + tid * rsize[0], &lrw,
        			iwork + tid * isize[0], &liw, myjex, &jt, &(common[tid]));

        	g_t[tid] = t[threadIdx.x];
        	g_tout[tid] = tout[threadIdx.x];
        	g_istate[tid] = istate[threadIdx.x];
        	memcpy(g_y+tid*neq, y+threadIdx.x*neq, sizeof(double)*neq);

        }
    }