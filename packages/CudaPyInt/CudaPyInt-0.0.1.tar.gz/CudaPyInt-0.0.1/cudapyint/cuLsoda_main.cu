#include "hip/hip_runtime.h"
    extern "C"{
    
        __device__ myFex myfex;
        __device__ myJex myjex;
        
        __global__ void init_common(){
            int tid = blockDim.x * blockIdx.x + threadIdx.x;
            cuLsodaCommonBlockInit( &(common[tid]) );
        }
        
        __global__ void cuLsoda(int *neq, double *y, double *t, double *tout, int *itol, 
                    double *rtol, double *atol, int *itask, int *istate, int *iopt, 
                                double *rwork, int *lrw, int *iwork, int *liw, int *jt, int *isize, int *rsize){
                                
            int tid = blockDim.x * blockIdx.x + threadIdx.x;
            
            dlsoda_(myfex, neq, y+tid*neq[0], t+tid, tout+tid, itol, rtol, atol, itask, 
                istate+tid, iopt, rwork+tid*rsize[0], lrw, iwork+tid*isize[0], liw, myjex, jt, &(common[tid]) );
        }
    }