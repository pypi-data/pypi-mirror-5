#include "hip/hip_runtime.h"
    extern "C"{
    
        __device__ myFex myfex;
        __device__ myJex myjex;
        
        __global__ void init_common(){
            int tid = blockDim.x * blockIdx.x + threadIdx.x;
            cuLsodaCommonBlockInit( &(common[tid]) );
        }
        
        __global__ void cuLsoda(int *neq, double *g_y, double *t, double *tout, int *itol, 
        			double *rtol, double *atol, int *itask, int *istate, int *iopt, 
        			double *g_rwork, int *lrw, int *iwork, int *liw, int *jt, int *isize, int *rsize) {

            int tid = blockIdx.x;

            dlsoda_(myfex, neq, g_y+tid*neq[0], t+tid, tout+tid, itol, rtol, atol, itask,
                istate+tid, iopt, g_rwork+tid*rsize[0], lrw, iwork+tid*isize[0], liw, myjex, jt, &(common[tid]) );

        }
    }