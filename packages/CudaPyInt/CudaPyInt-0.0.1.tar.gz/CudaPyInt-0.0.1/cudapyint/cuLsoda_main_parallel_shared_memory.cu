#include "hip/hip_runtime.h"
    extern "C"{
    
        __device__ myFex myfex;
        __device__ myJex myjex;
        
        __global__ void init_common(){
            int tid = blockDim.x * blockIdx.x + threadIdx.x;
            cuLsodaCommonBlockInit( &(common[tid]) );
        }
        
        __global__ void cuLsoda(int *g_neq, double *g_y, double *g_t,
        		double *g_tout, int *g_itol, double *g_rtol, double *g_atol, int *g_itask,
        		int *g_istate, int *g_iopt, double *g_rwork, int *g_lrw, int *g_iwork,
        		int *g_liw, int *g_jt, int *isize, int *rsize) {

        	__shared__ int neq;
        	__shared__ int itol;
        	__shared__ double rtol;
        	__shared__ int itask;
        	__shared__ int iopt;
        	__shared__ int lrw;
        	__shared__ int liw;
        	__shared__ int jt;
        	__shared__ double t;
        	__shared__ double tout;
        	__shared__ int istate;

        	__shared__ double y[NEQ];
        	__shared__ double atol[NEQ];
        	__shared__ int iwork[20+NEQ];
            if(threadIdx.x==0){
        		neq = g_neq[0];
        		itol = g_itol[0];
        		rtol = g_rtol[0];
        		itask = g_itask[0];
        		iopt = g_iopt[0];
        		lrw = g_lrw[0];
        		liw = g_liw[0];
        		jt = g_jt[0];

        		t = g_t[blockIdx.x];
        		tout = g_tout[blockIdx.x];
        		istate = g_istate[blockIdx.x];

        		memcpy(y, g_y+blockIdx.x*neq, sizeof(double)*neq);
        		memcpy(atol, g_atol, sizeof(double)*neq);
        		memcpy(iwork, g_iwork+blockIdx.x*isize[0], sizeof(int)*isize[0]);
            }
        	__syncthreads();

            dlsoda_(myfex, &neq, y, &t, &tout, &itol, &rtol, atol, &itask,
                &istate, &iopt, g_rwork+blockIdx.x*rsize[0], &lrw, iwork, &liw, myjex, &jt, &(common[blockIdx.x]) );

            if(threadIdx.x==0){
            	g_istate[blockIdx.x] = istate;
            	memcpy(g_y+blockIdx.x*neq, y, sizeof(double)*neq);
            	memcpy(g_iwork+blockIdx.x*isize[0], iwork, sizeof(int)*isize[0]);
            }
        //    __syncthreads();
        }
    }