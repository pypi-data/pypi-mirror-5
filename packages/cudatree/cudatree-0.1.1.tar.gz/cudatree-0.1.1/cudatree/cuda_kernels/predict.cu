#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define IDX_DATA_TYPE %s
#define SAMPLE_DATA_TYPE %s
#define LABEL_DATA_TYPE %s

__global__ void predict(IDX_DATA_TYPE *left_child_arr,
                        IDX_DATA_TYPE *right_child_arr,
                        uint16_t *feature_array,
                        float *threshold_array,
                        LABEL_DATA_TYPE *value_array,
                        SAMPLE_DATA_TYPE *predict_array,
                        LABEL_DATA_TYPE *predict_res,
                        int n_features,
                        int n_nodes
                        ){
  int offset = blockIdx.x * n_features;
  int idx = 0; 
  
  while(true){
    IDX_DATA_TYPE left_idx = left_child_arr[idx];
    IDX_DATA_TYPE right_idx = right_child_arr[idx];
    
    if(left_idx == 0 || right_idx == 0){
      //Means it's on leaf.
      predict_res[blockIdx.x] = value_array[idx];
      return;
    }
    
    float threshold = threshold_array[idx]; 
    uint16_t feature_idx = feature_array[idx];
    
    if(predict_array[offset + feature_idx] < threshold)
      idx = left_idx;
    else 
      idx = right_idx;
  }

}









