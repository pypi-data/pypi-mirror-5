#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>

#define THREADS_PER_BLOCK %d
#define MAX_NUM_LABELS %d
#define LABEL_DATA_TYPE %s
#define COUNT_DATA_TYPE %s
#define IDX_DATA_TYPE %s

__global__ void count_total(
                          IDX_DATA_TYPE *sorted_indices_1,
                          IDX_DATA_TYPE *sorted_indices_2,
                          LABEL_DATA_TYPE *labels,
                          COUNT_DATA_TYPE *label_total,
                          uint8_t *si_idx,
                          IDX_DATA_TYPE *begin_stop_idx,
                          IDX_DATA_TYPE *subset_indices,
                          int max_features
                          ){
  
  __shared__ IDX_DATA_TYPE *p_sorted_indices;
  __shared__ IDX_DATA_TYPE shared_start_idx;
  __shared__ IDX_DATA_TYPE shared_stop_idx;
  __shared__ COUNT_DATA_TYPE shared_count[MAX_NUM_LABELS];
  __shared__ LABEL_DATA_TYPE shared_labels[THREADS_PER_BLOCK];

  for(int i = threadIdx.x; i < MAX_NUM_LABELS; i += blockDim.x)
    shared_count[i] = 0;
  
  if(threadIdx.x == 0){
    shared_start_idx = begin_stop_idx[2 * blockIdx.x];
    shared_stop_idx = begin_stop_idx[2 * blockIdx.x + 1];
    
    uint8_t reg_si_idx = si_idx[blockIdx.x];
    if(reg_si_idx == 0)
      p_sorted_indices = sorted_indices_1;
    else 
      p_sorted_indices = sorted_indices_2;
  }
   
  __syncthreads();

  IDX_DATA_TYPE n_samples = shared_stop_idx - shared_start_idx;

  for(int i =  threadIdx.x; i < n_samples; i += blockDim.x){
    shared_labels[threadIdx.x] = labels[p_sorted_indices[shared_start_idx + i]];
    
    __syncthreads();
    
    if(threadIdx.x == 0){
      IDX_DATA_TYPE stop_pos = (i + blockDim.x  < n_samples)? blockDim.x : n_samples - i;

      for(int t = 0; t < stop_pos; ++t)
        shared_count[shared_labels[t]]++;
    }
    
    __syncthreads();
  }

  if(threadIdx.x == 0)
    for(int i = 0; i < MAX_NUM_LABELS; i++)
      label_total[blockIdx.x * MAX_NUM_LABELS + i] = shared_count[i];
}
 








