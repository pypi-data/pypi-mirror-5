#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>

#define THREADS_PER_BLOCK %d
#define MAX_NUM_LABELS %d
#define LABEL_DATA_TYPE %s
#define COUNT_DATA_TYPE %s
#define IDX_DATA_TYPE %s

__global__ void count_total(
                        IDX_DATA_TYPE *sorted_indices,
                        LABEL_DATA_TYPE *labels, 
                        COUNT_DATA_TYPE *label_total,
                        int n_samples
                        ){
   
  __shared__ COUNT_DATA_TYPE shared_count[MAX_NUM_LABELS];
  __shared__ LABEL_DATA_TYPE shared_labels[THREADS_PER_BLOCK]; 
  IDX_DATA_TYPE stop_pos;
  
  for(uint16_t i = threadIdx.x; i < MAX_NUM_LABELS; i += blockDim.x)
    shared_count[i] = 0;
  
  for(IDX_DATA_TYPE i = 0; i < n_samples; i += blockDim.x){
    IDX_DATA_TYPE idx = i + threadIdx.x;
    if(idx < n_samples)
      shared_labels[threadIdx.x] = labels[sorted_indices[idx]];
    
    __syncthreads();

    if(threadIdx.x == 0){
      stop_pos = (i + blockDim.x < n_samples)? blockDim.x : n_samples - i;

      for(IDX_DATA_TYPE t = 0; t < stop_pos; ++t)
        shared_count[shared_labels[t]]++;
    } 

    __syncthreads();
  }
  
  for(uint16_t i =  threadIdx.x; i < MAX_NUM_LABELS; i += blockDim.x)
    label_total[i] = shared_count[i];
}
