#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define IDX_DATA_TYPE %s
#define THREADS_PER_BLOCK %s

#define WARP_SIZE 32
#define WARP_MASK 0x1f

texture<char, 1> tex_mark;

__global__ void scan_reshuffle(
                          uint8_t* mark_table,
                          uint8_t* si_idx,
                          IDX_DATA_TYPE* sorted_indices_1,
                          IDX_DATA_TYPE* sorted_indices_2,
                          uint32_t* begin_end_idx,
                          IDX_DATA_TYPE* split,
                          uint32_t n_features,
                          uint32_t stride
                          ){  
  
  __shared__ IDX_DATA_TYPE last_sum;
   
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 300
  uint16_t lane_id = threadIdx.x & WARP_MASK;
  uint16_t warp_id = threadIdx.x / WARP_SIZE;
  __shared__ IDX_DATA_TYPE shared_pos_table[THREADS_PER_BLOCK / WARP_SIZE];
#else
  __shared__ IDX_DATA_TYPE shared_pos_table[THREADS_PER_BLOCK];
#endif 
  
  IDX_DATA_TYPE reg_start_idx = begin_end_idx[2 * blockIdx.x];
  IDX_DATA_TYPE reg_stop_idx = begin_end_idx[2 * blockIdx.x + 1];
  IDX_DATA_TYPE reg_split_idx = split[blockIdx.x];
  IDX_DATA_TYPE n;
  
  
  if(reg_split_idx == reg_stop_idx)
    return;
  

  IDX_DATA_TYPE *p_sorted_indices_in;
  IDX_DATA_TYPE *p_sorted_indices_out;

  if(si_idx[blockIdx.x] == 0){
    p_sorted_indices_in = sorted_indices_1;
    p_sorted_indices_out = sorted_indices_2;
  }else{
    p_sorted_indices_in = sorted_indices_2;
    p_sorted_indices_out = sorted_indices_1;
  }
  
  for(uint16_t shuffle_feature_idx = blockIdx.y; shuffle_feature_idx < n_features; shuffle_feature_idx += gridDim.y){
    uint32_t offset = shuffle_feature_idx * stride;

    if(threadIdx.x == 0)
      last_sum = 0;

    for(IDX_DATA_TYPE i = reg_start_idx; i < reg_stop_idx; i += blockDim.x){
      uint8_t side = 0;
      IDX_DATA_TYPE idx = i + threadIdx.x;
      IDX_DATA_TYPE reg_pos;
      IDX_DATA_TYPE si_idx; 
      
      if(idx < reg_stop_idx){
        si_idx = p_sorted_indices_in[offset + idx];
        side = tex1Dfetch(tex_mark, si_idx);
      }

      reg_pos = side;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 300

      for(uint16_t s = 1; s < WARP_SIZE; s *= 2){
        n = __shfl_up((int)reg_pos, s);
        if(lane_id >= s)
          reg_pos += n;
      }

      if(lane_id == WARP_SIZE - 1)
        shared_pos_table[warp_id] = reg_pos;
     
      __syncthreads();
     
      if(threadIdx.x == 0)
        for(uint16_t l = 1; l < blockDim.x / WARP_SIZE - 1; ++l)
          shared_pos_table[l] += shared_pos_table[l-1];

      __syncthreads();
      
      if(warp_id > 0)
        reg_pos += shared_pos_table[warp_id - 1];
      
      reg_pos += last_sum; 

#else
      shared_pos_table[threadIdx.x] = side;
      
      __syncthreads();

      for(uint16_t s = 1; s < blockDim.x; s *= 2){
        if(threadIdx.x >= s)
          n = shared_pos_table[threadIdx.x - s];
        else
          n = 0;
        __syncthreads();
        shared_pos_table[threadIdx.x] += n;
        __syncthreads();
      }
      
      reg_pos = shared_pos_table[threadIdx.x] + last_sum;
#endif

      if(idx < reg_stop_idx){
        IDX_DATA_TYPE out_pos = (side == 1)? reg_start_idx + reg_pos - 1 : reg_split_idx + 1 + idx - reg_start_idx - reg_pos;
        p_sorted_indices_out[offset + out_pos] = si_idx;   
      }

      __syncthreads();

      if(threadIdx.x == blockDim.x - 1)
        last_sum = reg_pos;
    }
     
    __syncthreads();
  }

}

