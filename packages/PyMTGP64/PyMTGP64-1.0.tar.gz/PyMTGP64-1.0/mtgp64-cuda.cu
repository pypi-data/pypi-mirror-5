#include "hip/hip_runtime.h"
/* \file mtgp64-cuda.cu
   \author R. Samadi (LESIA - Observatoire de Paris)
  
  This source file is part of the Python module PyMTGP64. 
  
  It is the CUDA implementation of MTGP, the Mersenne Twister for 
  Graphic Processors by Mutsuo Saito and Makoto Matsumoto (Hiroshima University).
  It provides random generators for uniform, Normal and Poisson distributions. 
  Only 64-bit floating numbers are generated.

  This source file was adapted from the original file named mtgp64-cuda.cu 
  and developped by Mutsuo Saito and Makoto Matsumoto (see copyright below).

Copyright (c) 2013 by R. Samadi (LESIA - Observatoire de Paris)

This is a free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
 
This software is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
 
You should have received a copy of the GNU General Public License
along with this code.  If not, see <http://www.gnu.org/licenses/>.
 


Copyright (c) 2009, 2010 Mutsuo Saito, Makoto Matsumoto and Hiroshima
University.
Copyright (c) 2011, 2012 Mutsuo Saito, Makoto Matsumoto, Hiroshima
University and University of Tokyo.
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above
      copyright notice, this list of conditions and the following
      disclaimer in the documentation and/or other materials provided
      with the distribution.
    * Neither the name of the Hiroshima University, The Uinversity
      of Tokyo nor the names of its contributors may be used to
      endorse or promote products derived from this software without
      specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <stdlib.h>
#include <math.h>

#include "sample-cuda.h"
#include "mtgp-util.cuh"
//#include "mtgp-print.h"
#include "mtgp64-fast.h"
#include "mtgp64-const.h"
#include<sm_13_double_functions.h>

#define PI 3.14159265358979311599796346854418516

extern const int mtgpdc_params_11213_num;
extern mtgp64_params_fast_t mtgp64dc_params_fast_11213[];


/**
 * kernel I/O
 * This structure must be initialized before first use.
 */
struct mtgp64_kernel_status_t {
    uint64_t status[MTGPDC_N];
};

/*
 * Generator Parameters.
 */
__constant__ uint32_t pos_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t sh1_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t sh2_tbl[BLOCK_NUM_MAX];
__constant__ uint32_t mask[2];
__constant__ uint32_t param_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t temper_tbl[BLOCK_NUM_MAX][TBL_SIZE];
__constant__ uint32_t double_temper_tbl[BLOCK_NUM_MAX][TBL_SIZE];

/**
 * Shared memory
 * The generator's internal status vector.
 */
__shared__ uint32_t status[2][LARGE_SIZE]; /* 512 * 3 elements, 12288 bytes. */

/**
 * The function of the recursion formula calculation.
 *
 * @param[out] RH 32-bit MSBs of output
 * @param[out] RL 32-bit LSBs of output
 * @param[in] X1H MSBs of the farthest part of state array.
 * @param[in] X1L LSBs of the farthest part of state array.
 * @param[in] X2H MSBs of the second farthest part of state array.
 * @param[in] X2L LSBs of the second farthest part of state array.
 * @param[in] YH MSBs of a part of state array.
 * @param[in] YL LSBs of a part of state array.
 * @param[in] bid block id.
 */
__device__ void para_rec(uint32_t *RH,
			 uint32_t *RL,
			 uint32_t X1H,
			 uint32_t X1L,
			 uint32_t X2H,
			 uint32_t X2L,
			 uint32_t YH,
			 uint32_t YL,
			 int bid) {
    uint32_t XH = (X1H & mask[0]) ^ X2H;
    uint32_t XL = (X1L & mask[1]) ^ X2L;
    uint32_t MAT;

    XH ^= XH << sh1_tbl[bid];
    XL ^= XL << sh1_tbl[bid];
    YH = XL ^ (YH >> sh2_tbl[bid]);
    YL = XH ^ (YL >> sh2_tbl[bid]);
    MAT = param_tbl[bid][YL & 0x0f];
    *RH = YH ^ MAT;
    *RL = YL;
}

/**
 * The tempering function.
 *
 * @param[in] VH MSBs of the output value should be tempered.
 * @param[in] VL LSBs of the output value should be tempered.
 * @param[in] TL LSBs of the tempering helper value.
 * @param[in] bid block id.
 * @return[in] the tempered value.
 */
__device__ uint64_t temper(uint32_t VH,
			   uint32_t VL,
			   uint32_t TL,
			   int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = temper_tbl[bid][TL & 0x0f];
    VH ^= MAT;
    r = ((uint64_t)VH << 32) | VL;
    return r;
}

/**
 * The tempering and converting function.
 * By using the preset-ted table, converting to IEEE format
 * and tempering are done simultaneously.
 *
 * @param[in] VH MSBs of the output value should be tempered.
 * @param[in] VL LSBs of the output value should be tempered.
 * @param[in] TL LSBs of the tempering helper value.
 * @param[in] bid block id.
 * @return the tempered and converted value.
 */
__device__ uint64_t temper_double(uint32_t VH,
				  uint32_t VL,
				  uint32_t TL,
				  int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = double_temper_tbl[bid][TL & 0x0f];
    r = ((uint64_t)VH << 32) | VL;
    r = (r >> 12) ^ ((uint64_t)MAT << 32);
    return r;
}
/**
 * The tempering and converting function.
 * By using the preset-ted table, converting to IEEE format
 * and tempering are done simultaneously.
 *
 * @param[in] VH MSBs of the output value should be tempered.
 * @param[in] VL LSBs of the output value should be tempered.
 * @param[in] TL LSBs of the tempering helper value.
 * @param[in] bid block id.
 * @return the tempered and converted value.
 */
__device__ uint64_t temper_double_open(uint32_t VH,
				  uint32_t VL,
				  uint32_t TL,
				  int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = double_temper_tbl[bid][TL & 0x0f];
    r = ((uint64_t)VH << 32) | VL;
    r = (  ( r >> 12 ) ^ ((uint64_t)MAT << 32) ) | 1 ; 
    return r;
}

/**
 * Read the internal state vector from kernel I/O data, and
 * put them into shared memory.
 *
 * @param[out] status shared memory.
 * @param[in] d_status kernel I/O data
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_read(uint32_t status[2][LARGE_SIZE],
			    const mtgp64_kernel_status_t *d_status,
			    int bid,
			    int tid) {
    uint64_t x;

    x = d_status[bid].status[tid];
    status[0][LARGE_SIZE - N + tid] = x >> 32;
    status[1][LARGE_SIZE - N + tid] = x & 0xffffffff;
    if (tid < N - THREAD_NUM) {
	x = d_status[bid].status[THREAD_NUM + tid];
	status[0][LARGE_SIZE - N + THREAD_NUM + tid] = x >> 32;
	status[1][LARGE_SIZE - N + THREAD_NUM + tid] = x & 0xffffffff;
    }
    __syncthreads();
}

/**
 * Read the internal state vector from shared memory, and
 * write them into kernel I/O data.
 *
 * @param[out] status shared memory.
 * @param[in] d_status kernel I/O data
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_write(mtgp64_kernel_status_t *d_status,
			     const uint32_t status[2][LARGE_SIZE],
			     int bid,
			     int tid) {
    uint64_t x;

    x = (uint64_t)status[0][LARGE_SIZE - N + tid] << 32;
    x = x | status[1][LARGE_SIZE - N + tid];
    d_status[bid].status[tid] = x;
    if (tid < N - THREAD_NUM) {
	x = (uint64_t)status[0][4 * THREAD_NUM - N + tid] << 32;
	x = x | status[1][4 * THREAD_NUM - N + tid];
	d_status[bid].status[THREAD_NUM + tid] = x;
    }
    __syncthreads();
}

/**
 * kernel function.
 * This function generates 64-bit unsigned integers in d_data
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output
 * @param[in] size number of output data requested.
 */
__global__ void mtgp64_uint64_kernel(mtgp64_kernel_status_t* d_status,
				     uint64_t* d_data, int size) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {

#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][LARGE_SIZE - N + tid]:%08x\n",
		   status[0][LARGE_SIZE - N + tid]);
	    printf("status[1][LARGE_SIZE - N + tid]:%08x\n",
		   status[1][LARGE_SIZE - N + tid]);
	    printf("status[0][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + 1]);
	    printf("status[1][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + 1]);
	    printf("status[0][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + pos]);
	    printf("status[1][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + pos]);
	    printf("sh1:%d\n", sh1_tbl[bid]);
	    printf("sh2:%d\n", sh2_tbl[bid]);
	    printf("high_mask:%08x\n", mask[0]);
	    printf("low_mask:%08x\n", mask[1]);
	    for (int j = 0; j < 16; j++) {
		printf("tbl[%d]:%08x\n", j, param_tbl[0][j]);
	    }
	}
#endif
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][tid]:%08x\n",	status[0][tid]);
	    printf("status[1][tid]:%08x\n",	status[1][tid]);
	}
#endif
	o = temper(YH,
		   YL,
		   status[1][LARGE_SIZE - N + tid + pos - 1],
		   bid);
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("o:%016" PRIx64 "\n", o);
	}
#endif
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
		   bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
		   bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * kernel function.
 * This function generates double precision floating point numbers 
 * uniformly distributed in the range ]0,1[  (opened interval).
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output. IEEE double precision format.
 * @param[in] size number of output data requested.
 */
__global__ void mtgp64_double_kernel(mtgp64_kernel_status_t* d_status,
				     double* d_data, int size)
{

    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
	o = temper_double_open(YH,
			  YL,
			  status[1][LARGE_SIZE - N + tid + pos - 1],
			  bid);
	((uint64_t *) d_data)[size * bid + i + tid] = o  ;
	d_data[size * bid + i + tid] -= 1. ;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper_double_open(
	    YH,
	    YL,
	    status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
	    bid);
	((uint64_t *) d_data)[size * bid + THREAD_NUM + i + tid] = o  ;
	d_data[size * bid + THREAD_NUM + i + tid] -= 1. ;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper_double_open(YH,
			  YL,
			  status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
			  bid);
	((uint64_t *)d_data)[size * bid + 2 * THREAD_NUM + i + tid] = o  ;
	d_data[size * bid + 2 * THREAD_NUM + i + tid] -= 1. ;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/*
	void mtgp64_double_normal_kernel(mtgp64_kernel_status_t* d_status, double * d_data1,  double * d_data2, int size)

	Computes two sets of normally distributed values from two given sets of uniformly distributed values.


  INPUTS:
  mtgp64_kernel_status_t * d_status : pointer to the structure containing the state vectors
  int block_size : number of values per block

  INPUTS/OUTPUTS:
  double d_data1[n], d_data2[n]:  contain as inputs two given sets of uniformly distributed values, and as outpus the random numbers distributed according to the Normal distribution.  The size of the arrays are  n = block_size * block_number


 R. Samadi
*/

__global__ void mtgp64_double_normal_kernel(mtgp64_kernel_status_t* d_status, double * d_data1,  double * d_data2, int size)

{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    double r1,r2 ;
    int j ;

    //  loop for the calculation of the normally distributed values
    for (int i = 0; i < size; i += THREAD_NUM ) {
      j = size * bid + i + tid ;
      r1 = sqrt(-2.*log(d_data1[j]))  ;
      r2 = d_data2[j] ; 
      d_data1[j] = r1 * cos(2.* PI * r2) ;
      d_data2[j] = r1 * sin(2.* PI * r2) ;
    }
}

/* log-gamma function to support some of these distributions. The 
 * algorithm comes from SPECFUN by Shanjie Zhang and Jianming Jin and their
 * book "Computation of Special Functions", 1996, John Wiley & Sons, Inc.
 */
__device__ double loggam(double x)
{
    double x0, x2, xp, gl, gl0;
    long k, n;
    
    double a[10] = {8.333333333333333e-02,-2.777777777777778e-03,
         7.936507936507937e-04,-5.952380952380952e-04,
         8.417508417508418e-04,-1.917526917526918e-03,
         6.410256410256410e-03,-2.955065359477124e-02,
         1.796443723688307e-01,-1.39243221690590e+00};
    x0 = x;
    n = 0;
    if ((x == 1.0) || (x == 2.0))
    {
        return 0.0;
    }
    else if (x <= 7.0)
    {
        n = (long)(7 - x);
        x0 = x + n;
    }
    x2 = 1.0/(x0*x0);
    xp = 2*PI;
    gl0 = a[9];
    for (k=8; k>=0; k--)
    {
        gl0 *= x2;
        gl0 += a[k];
    }
    gl = gl0/x0 + 0.5*log(xp) + (x0-0.5)*log(x0) - x0;
    if (x <= 7.0)
    {
        for (k=1; k<=n; k++)
        {
            gl -= log(x0-1.0);
            x0 -= 1.0;
        }
    }
    return gl;
}


/*
  void mtgp64_poisson_kernel(mtgp64_kernel_status_t* d_status, long * d_data, int block_size , double lam)
  
  Compute Poisson-distributed random numbers.

  
  INPUTS:
  mtgp64_kernel_status_t * d_status : pointer to the structure containing the state vectors
  int block_size : number of values per block
  double lam : mean value ('lambda') of the Poisson distribution

  OUTPUTS:
  
  double d_data[n]:  the random numbers distributed according to the Poisson distribution.  The size of the array is assumed to be  n = block_size * block_number

  Adapted from rk_poisson_mult and rk_poisson_ptrsand by Robert Kern (robert.kern@gmail.com)

  R. Samadi
*/
__global__ void mtgp64_poisson_kernel(mtgp64_kernel_status_t* d_status, long * d_data, int block_size , double lam)
{

  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  int pos = pos_tbl[bid];
  uint32_t YH;
  uint32_t YL;
  uint64_t o;
  long k;
  double U, V, slam, loglam, a, b, invalpha, vr, us;
  double prod, enlam;
  int fac[3] = {3,4,2} ;
  int j , m  ; 
  bool cont ; 
  bool notall ;
  __shared__ bool flag[THREAD_NUM] ; 

  if (lam>=10) {
    slam = sqrt(lam);
    loglam = log(lam);
    b = 0.931 + 2.53*slam;
    a = -0.059 + 0.02483*b;
    invalpha = 1.1239 + 1.1328/(b-3.4);
    vr = 0.9277 - 3.6224/(b-2);
  }
  else enlam = exp(-lam);

  // copy status data from global memory to shared memory.
  status_read(status, d_status, bid, tid);

  // main loop
  for (int i = 0; i < block_size; i += THREAD_NUM ) {
    m = block_size * bid + tid  + i ; 
    if( lam<10) {
      k = 0;
      prod = 1.0;
    }
    j = 0 ;
    cont = 1 ;
    notall = 1; 
    flag[tid] = 0 ; 
    while (cont || notall) {
      // generating the random number U
      para_rec(&YH,
	       &YL,
	       status[0][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
	       status[1][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
	       status[0][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
	       status[1][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
	       status[0][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
	       status[1][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
	       bid);
      status[0][tid + j*THREAD_NUM] = YH;
      status[1][tid + j*THREAD_NUM] = YL;
	
      o = temper_double_open(
			YH,
			YL,
			status[1][(fac[j]*THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
			bid);
      * ( ( uint64_t * )  (&U) ) = o ; 
      U = 2. - U  ;
      j = (j +1) % 3 ;
      __syncthreads();

      if( lam >= 10) {
	// generating the random number V
	para_rec(&YH,
		 &YL,
		 status[0][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + j*THREAD_NUM] = YH;
	status[1][tid + j*THREAD_NUM] = YL;
	o = temper_double_open(
			  YH,
			  YL,
			  status[1][(fac[j]*THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
			  bid);
	* ( ( uint64_t * )  (&V) ) = o ; 
        j = (j +1) % 3 ;
	__syncthreads();
	if (cont) {
	V = 2. - V  ; 
	U = U - 0.5;
        us = 0.5 - fabs(U);
        k = (long)floor((2*a/us + b)*U + lam + 0.43);
        if ((us >= 0.07) && (V <= vr))
	  {
	    d_data[m] = k ;
	    cont = 0 ;
	  }
        else if ((k < 0) ||
            ((us < 0.013) && (V > us)))
	  {
            cont = 1 ;
	  }
        else if ((log(V) + log(invalpha) - log(a/(us*us)+b)) <=
            (-lam + k*loglam - loggam(k+1)))
	  {
	    d_data[m] = k ; 
            cont = 0 ;
	  }
	} // end if (cont)
      } // end if( lam >=10)
      else if (lam >0.)
	{
	  if(cont) {
	  prod *= U;
	  if (prod > enlam)
	    {
	      k += 1;            
	      cont = 1 ;
	    }
	  else  {
		d_data[m] = k;
            	cont = 0 ;
		}
	 } // end if (cont)
	} //end else if ( lam >0)
      else {
	d_data[m] = 0 ; 
	cont = 0 ;
      }
     	if(  ! cont ) flag[tid] = 1 ;
	__syncthreads();
	notall = 0 ;
	for (int p=0 ; (p<THREAD_NUM) & (! notall) ; p++ ) notall = notall || (! flag[p] ) ; 
	__syncthreads();
    } // end while(cont)
  } // end for loop
    // write back status for next call
  status_write(d_status, status, bid, tid);
}

/*
  void mtgp64_poisson_multlam_kernel(mtgp64_kernel_status_t* d_status, double * d_data,  int block_size )

  Compute Poisson-distributed random numbers.

  While mtgp64_poisson_kernel works with a single value of 'lambda', this version considers multiple values of lambda (one for each generated random number).
  
  INPUTS:
  d_status : pointer to the structure containing the state vectors
  block_size : number of values per block

  INPUTS/OUTPUTS:
  
  double d_data[n]:  contains as input the lambda values (mean value of the Poisson distribution),
  and as output the random numbers distributed according to the Poisson distribution.  
  The size of the array is assumed to be  n = block_size * block_number


  Adapted from rk_poisson_mult and rk_poisson_ptrsand by Robert Kern (robert.kern@gmail.com)

  R. Samadi
*/
__global__ void mtgp64_poisson_multlam_kernel(mtgp64_kernel_status_t* d_status, double * d_data,  int block_size )
{
const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  int pos = pos_tbl[bid];
  uint32_t YH;
  uint32_t YL;
  uint64_t o;
  long k;
  double U, V, slam, loglam, a, b, invalpha, vr, us;
  double prod, enlam , lam ;
  int fac[3] = {3,4,2} ;
  int j , m , l  ; 
  bool cont ; 
  bool notall ;
  __shared__ bool flag[THREAD_NUM] ; 

  // copy status data from global memory to shared memory.
  status_read(status, d_status, bid, tid);

  // main loop
  for (int i = 0; i < block_size; i += THREAD_NUM ) {
    m = block_size * bid + tid  + i ; 
    lam = d_data[m] ;
    if( lam<10) {
      k = 0;
      prod = 1.0;
    }
    else {
       slam = sqrt(lam);
      loglam = log(lam);
      b = 0.931 + 2.53*slam;
      a = -0.059 + 0.02483*b;
      invalpha = 1.1239 + 1.1328/(b-3.4);
      vr = 0.9277 - 3.6224/(b-2);
    }
    j = 0 ;
    cont = 1 ;
    notall = 1; 
    flag[tid] = 0 ; 
    while (cont || notall) {
      // generating the random number U
      para_rec(&YH,
	       &YL,
	       status[0][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
	       status[1][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
	       status[0][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
	       status[1][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
	       status[0][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
	       status[1][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
	       bid);
      status[0][tid + j*THREAD_NUM] = YH;
      status[1][tid + j*THREAD_NUM] = YL;
	
      o = temper_double_open(
			YH,
			YL,
			status[1][(fac[j]*THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
			bid);
      * ( ( uint64_t * )  (&U) ) = o ; 
      U = 2. - U  ;
      j = (j +1) % 3 ;
      __syncthreads();
	// generating the random number V
	para_rec(&YH,
		 &YL,
		 status[0][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(fac[j]*THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(fac[j]*THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(fac[j]*THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + j*THREAD_NUM] = YH;
	status[1][tid + j*THREAD_NUM] = YL;
	o = temper_double_open(
			  YH,
			  YL,
			  status[1][(fac[j]*THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
			  bid);
	* ( ( uint64_t * )  (&V) ) = o ; 
        j = (j +1) % 3 ;
	__syncthreads();

	if (cont) {
      if( lam >= 10) {
	V = 2. - V  ; 
	U = U - 0.5;
        us = 0.5 - fabs(U);
        k = (long)floor((2*a/us + b)*U + lam + 0.43);
        if ((us >= 0.07) && (V <= vr))
	  {
	    d_data[m] = k ;
	    cont = 0 ;
	  }
        else if ((k < 0) ||
            ((us < 0.013) && (V > us)))
	  {
            cont = 1 ;
	  }
        else if ((log(V) + log(invalpha) - log(a/(us*us)+b)) <=
            (-lam + k*loglam - loggam(k+1)))
	  {
	    d_data[m] = k ; 
            cont = 0 ;
	  }
      } // end if( lam >=10)
      else if (lam >0.)
	{
	  l = 0 ;
	  while( (l<2) && (cont) ) {
	    if(l ==0)  prod *= U;
	    else prod *= V;
	    if (prod > enlam)
	      {
		k += 1;            
		cont = 1 ;
	      }
	    else  {
	      d_data[m] = k;
	      cont = 0 ;
	    }
	    l ++ ; 
	  } // end while( (l<2) && (cont) ) 
	} //end else if ( lam >0)
      else{
	d_data[m] = 0 ; 
	cont = 0 ;
 	} 
	}// end if (cont)
	if(  ! cont ) flag[tid] = 1 ;
	__syncthreads();
	notall = 0 ;
	for (int p=0 ; (p<THREAD_NUM) & (! notall) ; p++ ) notall = notall || (! flag[p] ) ; 
	__syncthreads();
    } // end while(cont)
  } // end for loop
    // write back status for next call
  status_write(d_status, status, bid, tid);
}



/**
 * This function initializes kernel I/O data and setups the seed of each block.
 * 
 * 
 * INPUTS:
 * d_status: output kernel I/O data.
 * params: MTGP64 parameters. needed for the initialization.
 * int block_num: the number of blocks
 * uint64_t seeds[block_num]: the block seeds
 * 
 */
extern "C" __host__ void make_kernel_data64(mtgp64_kernel_status_t *d_status,
			mtgp64_params_fast_t params[],
				 int block_num , uint64_t * seeds)
{
    mtgp64_kernel_status_t* h_status
	= (mtgp64_kernel_status_t *) malloc(
	    sizeof(mtgp64_kernel_status_t) * block_num);

    if (h_status == NULL) {
	printf("failure in allocating host memory for kernel I/O data.\n");
	exit(8);
    }
    for (int i = 0; i < block_num; i++) {
	mtgp64_init_state(&(h_status[i].status[0]), &params[i], seeds[i]);
    }
#if defined(DEBUG)
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[0]);
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[1]);
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[2]);
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[3]);
#endif
    ccudaMemcpy(d_status, h_status,
		sizeof(mtgp64_kernel_status_t) * block_num,
		hipMemcpyHostToDevice);
    free(h_status);
}




/**
 * This function sets constants in device memory.
 * @param[in] params input, MTGP64 parameters.
 */
__host__ void make_constant(const mtgp64_params_fast_t params[],
		   int block_num) {
    const int size1 = sizeof(uint32_t) * block_num;
    const int size2 = sizeof(uint32_t) * block_num * TBL_SIZE;
    uint32_t *h_pos_tbl;
    uint32_t *h_sh1_tbl;
    uint32_t *h_sh2_tbl;
    uint32_t *h_param_tbl;
    uint32_t *h_temper_tbl;
    uint32_t *h_double_temper_tbl;
    uint32_t *h_mask;
    h_pos_tbl = (uint32_t *)malloc(size1);
    h_sh1_tbl = (uint32_t *)malloc(size1);
    h_sh2_tbl = (uint32_t *)malloc(size1);
    h_param_tbl = (uint32_t *)malloc(size2);
    h_temper_tbl = (uint32_t *)malloc(size2);
    h_double_temper_tbl = (uint32_t *)malloc(size2);
    h_mask = (uint32_t *)malloc(sizeof(uint32_t) * 2);
    if (h_pos_tbl == NULL
	|| h_sh1_tbl == NULL
	|| h_sh2_tbl == NULL
	|| h_param_tbl == NULL
	|| h_temper_tbl == NULL
	|| h_double_temper_tbl == NULL
	|| h_mask == NULL
	) {
	printf("failure in allocating host memory for constant table.\n");
	exit(1);
    }
    h_mask[0] = params[0].mask >> 32;
    h_mask[1] = params[0].mask & 0xffffffffU;
    for (int i = 0; i < block_num; i++) {
	h_pos_tbl[i] = params[i].pos;
	h_sh1_tbl[i] = params[i].sh1;
	h_sh2_tbl[i] = params[i].sh2;
	for (int j = 0; j < TBL_SIZE; j++) {
	    h_param_tbl[i * TBL_SIZE + j] = params[i].tbl[j] >> 32;
	    h_temper_tbl[i * TBL_SIZE + j] = params[i].tmp_tbl[j] >> 32;
	    h_double_temper_tbl[i * TBL_SIZE + j]
		= params[i].dbl_tmp_tbl[j] >> 32;
	}
    }
    // copy from malloc area only
    ccudaMemcpyToSymbol(pos_tbl, h_pos_tbl, size1);
    ccudaMemcpyToSymbol(sh1_tbl, h_sh1_tbl, size1);
    ccudaMemcpyToSymbol(sh2_tbl, h_sh2_tbl, size1);
    ccudaMemcpyToSymbol(param_tbl, h_param_tbl, size2);
    ccudaMemcpyToSymbol(temper_tbl, h_temper_tbl, size2);
    ccudaMemcpyToSymbol(double_temper_tbl, h_double_temper_tbl, size2);
    ccudaMemcpyToSymbol(mask, &h_mask, sizeof(uint32_t) * 2);
    free(h_pos_tbl);
    free(h_sh1_tbl);
    free(h_sh2_tbl);
    free(h_param_tbl);
    free(h_temper_tbl);
    free(h_double_temper_tbl);
    free(h_mask);
}


/*
	Returns a number which is close to num_data and a mutliple of  LARGE_SIZE * block_num
*/

__host__ long get_num_unit( long num_data  , int block_num)
{
   long num_unit = LARGE_SIZE * block_num;
   int r;
   r = num_data % num_unit;
   if (r != 0) {
      num_unit = num_data + num_unit - r;
    }
   return num_unit ;
}


/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
extern "C"  __host__ uint64_t *  make_uint64_random(mtgp64_kernel_status_t* d_status,
			 long num_data, int block_num, int verbose = 0) {
    uint64_t* d_data;
    uint64_t* h_data;
    hipError_t e;
    float gputime;
    long num_unit = get_num_unit(num_data,block_num) ;
    hipEvent_t start;
    hipEvent_t end;

    ccudaMalloc((void**)&d_data, sizeof(uint64_t) * num_unit);
    /* CUT_SAFE_CALL(cutCreateTimer(&timer)); */
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);
    h_data = (uint64_t *) malloc(sizeof(uint64_t) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    /* CUT_SAFE_CALL(cutStartTimer(timer)); */
    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");	
	free(h_data) ; 
	exit(1);
    }

    /* kernel call */
    mtgp64_uint64_kernel<<< block_num, THREAD_NUM>>>(
	d_status, d_data, num_unit / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));	
	free(h_data) ; 
	exit(1);
    }
    /* CUT_SAFE_CALL(cutStopTimer(timer)); */
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaMemcpy(h_data, d_data, sizeof(uint64_t) * num_data,
		hipMemcpyDeviceToHost);
    /* gputime = cutGetTimerValue(timer); */
    if (verbose) {
      ccudaEventElapsedTime(&gputime, start, end);
//      print_uint64_array(h_data, num_data, block_num);
      printf("generated numbers: %d\n", num_data);
      printf("Processing time: %f (ms)\n", gputime);
      printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    }
    /* CUT_SAFE_CALL(cutDeleteTimer(timer)); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    //free memories
    // free(h_data);
    ccudaFree(d_data);
    return h_data  ;
}

/**
 * host function.
 * This function calls corresponding kernel function.
 * It returns a pointer to an array containing <num_data> random values
 * A NULL pointer is returned in case of failure, 
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
extern "C"  __host__ double * make_double_random(mtgp64_kernel_status_t* d_status,
			long num_data, int block_num , int verbose = 0 ) {
    double* d_data;
    double* h_data;
    hipError_t e;
    float gputime;
    long num_unit = get_num_unit(num_data,block_num) ;
    hipEvent_t start;
    hipEvent_t end;

    ccudaMalloc((void**)&d_data, sizeof(double) * num_unit);
    /* CUT_SAFE_CALL(cutCreateTimer(&timer)); */
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);
    h_data = (double *) malloc(sizeof(double) * num_data);
    if (h_data == NULL) {
	fprintf(stderr,"failure in allocating host memory for output data.\n");
	return 0 ;
    }
    /* CUT_SAFE_CALL(cutStartTimer(timer)); */
    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
	fprintf(stderr,"error has been occured before kernel call.\n");
	free(h_data) ; 
	return 0 ;
    }

    /* kernel call */
    mtgp64_double_kernel<<< block_num, THREAD_NUM >>>(
	d_status, d_data, num_unit / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	free(h_data) ; 
	return 0;
    }
    /* CUT_SAFE_CALL(cutStopTimer(timer)); */
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaMemcpy(h_data, d_data, sizeof(double) * num_data,
		hipMemcpyDeviceToHost);
    /* gputime = cutGetTimerValue(timer); */
    if (verbose) {			  
    ccudaEventElapsedTime(&gputime, start, end);

//    print_double_array(h_data, num_data, block_num);
    printf("Generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    }

    /* CUT_SAFE_CALL(cutDeleteTimer(timer)); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    //free memories
//    free(h_data);
    ccudaFree(d_data);
    return h_data ; 
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
extern "C"  __host__  int  make_double_normal_random(mtgp64_kernel_status_t* d_status,
			long num_data, int block_num ,  double * * h_data1,  double * *  h_data2, int verbose = 0 ) {
    double * d_data1;
    double * d_data2;
    hipError_t e;
    float gputime;
    long num_unit = get_num_unit(num_data,block_num) ;
    hipEvent_t start;
    hipEvent_t end;
    * h_data1 = NULL ; 
    * h_data2 = NULL ; 
    ccudaMalloc((void**)&d_data1, sizeof(double) * num_unit);
    ccudaMalloc((void**)&d_data2, sizeof(double) * num_unit);
    /* CUT_SAFE_CALL(cutCreateTimer(&timer)); */
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);
    * h_data1 = (double *) malloc(sizeof(double) * num_data);
    if (* h_data1 == NULL) {
	fprintf(stderr,"failure in allocating host memory for output data.\n");
	return 0 ;
    }
    * h_data2 = (double *) malloc(sizeof(double) * num_data);
    if (* h_data2 == NULL) {
	fprintf(stderr,"failure in allocating host memory for output data.\n");
	return 0 ;
    }
    /* CUT_SAFE_CALL(cutStartTimer(timer)); */
    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
	fprintf(stderr,"error has been occured before kernel call.\n");
	return 0 ;
    }

    /* kernel call */
    mtgp64_double_kernel<<< block_num, THREAD_NUM >>>(
	d_status, d_data1,  num_unit / block_num);
    hipDeviceSynchronize();

    /* kernel call */
    mtgp64_double_kernel<<< block_num, THREAD_NUM >>>(
        d_status, d_data2,  num_unit / block_num);
    hipDeviceSynchronize();

    /* kernel call */
    mtgp64_double_normal_kernel<<< block_num, THREAD_NUM >>>(
        d_status, d_data1, d_data2, num_unit / block_num);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	return 0;
    }
    /* CUT_SAFE_CALL(cutStopTimer(timer)); */
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaMemcpy(*h_data1, d_data1, sizeof(double) * num_data,
		hipMemcpyDeviceToHost);
    ccudaMemcpy(*h_data2, d_data2, sizeof(double) * num_data,
		hipMemcpyDeviceToHost);
    
    /* gputime = cutGetTimerValue(timer); */
    if (verbose) {			  
    ccudaEventElapsedTime(&gputime, start, end);

//    print_double_array(*h_data1, num_data, block_num);
    printf("Generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    }

    /* CUT_SAFE_CALL(cutDeleteTimer(timer)); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    //free memories
//    free(h_data);
    ccudaFree(d_data1);
    ccudaFree(d_data2);
    return 1 ; 
}


/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 * @param[in] lambda : mean value of the Poisson distribution
 */

extern "C"  __host__  long *  make_poisson_random(mtgp64_kernel_status_t* d_status, long num_data, int block_num, double lambda , int verbose = 0) {
    long* d_data;
    long* h_data;
    hipError_t e;
    float gputime;
    long num_unit = get_num_unit(num_data,block_num) ;
    hipEvent_t start;
    hipEvent_t end;

    ccudaMalloc((void**)&d_data, sizeof(long) * num_unit);
    /* CUT_SAFE_CALL(cutCreateTimer(&timer)); */
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);
    h_data = (long *) malloc(sizeof(long) * num_data);
    if (h_data == NULL) {
        printf("failure in allocating host memory for output data.\n");
        exit(1);
    }
    /* CUT_SAFE_CALL(cutStartTimer(timer)); */
    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
        printf("error has been occured before kernel call.\n");
        free(h_data) ;
        exit(1);
    }
    /* kernel call */
    mtgp64_poisson_kernel<<< block_num, THREAD_NUM>>>(d_status, d_data, num_unit / block_num,lambda);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
        printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
        free(h_data) ;
        exit(1);
    }
    /* CUT_SAFE_CALL(cutStopTimer(timer)); */
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaMemcpy(h_data, d_data, sizeof(long) * num_data,
                hipMemcpyDeviceToHost);
    /* gputime = cutGetTimerValue(timer); */
    if (verbose) {
      ccudaEventElapsedTime(&gputime, start, end);
 //     print_uint64_array(h_data, num_data, block_num);
      printf("generated numbers: %d\n", num_data);
      printf("Processing time: %f (ms)\n", gputime);
      printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    }
    /* CUT_SAFE_CALL(cutDeleteTimer(timer)); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    //free memories
    // free(h_data);
    ccudaFree(d_data);
    return h_data  ;
}


/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] lam[num_data]  the lambda values 
 *
 */

extern "C"  __host__  double *  make_poisson_multlam_random(mtgp64_kernel_status_t* d_status , long num_data, int block_num, double * lam,  int verbose = 0) {
    double * d_data ;
    double * h_data ;
    double * x ;
    hipError_t e;
    float gputime;
    long num_unit = get_num_unit(num_data,block_num) ;
    hipEvent_t start;
    hipEvent_t end;

    // allocation of the device memory
    ccudaMalloc((void**)&d_data, sizeof(double) * num_unit);

    // allocation of the output array 
    x = (double *) malloc(sizeof(double) * num_data);
    if (x == NULL) {
        printf("failure in allocating host memory for output data.\n");
        exit(1);
    }

    // the lambda values must be copied into d_data
    // we fist copy lam into a tempory array of size num_unit 
    // and fill the additional elements by zero.
    h_data =  (double *) malloc(sizeof(double) * num_unit);
    if (h_data== NULL) {
        printf("failure in allocating host memory\n");
        exit(1);
    }
    memcpy(h_data,lam,sizeof(double) * num_data);
    // we fill the additional elements by zero
    for (int i=num_data ; i < num_unit ; i++) h_data[i] = 0. ;

    // we finally copy h_data into the device data
    ccudaMemcpy(d_data, h_data, sizeof(double) * num_unit,hipMemcpyHostToDevice);
    free(h_data) ;

    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    if (hipGetLastError() != hipSuccess) {
        printf("error has been occured before kernel call.\n");
        exit(1);
    }
    /* kernel call */
    mtgp64_poisson_multlam_kernel<<< block_num, THREAD_NUM>>>(d_status, d_data, num_unit / block_num) ;
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
        printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
        exit(1);
    }
    /* CUT_SAFE_CALL(cutStopTimer(timer)); */
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaMemcpy(x, d_data, sizeof(double) * num_data,hipMemcpyDeviceToHost);

    /* gputime = cutGetTimerValue(timer); */
    if (verbose) {
      ccudaEventElapsedTime(&gputime, start, end);
      printf("generated numbers: %d\n", num_data);
      printf("Processing time: %f (ms)\n", gputime);
      printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    }
    /* CUT_SAFE_CALL(cutDeleteTimer(timer)); */
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    //free memories
    ccudaFree(d_data);
    return x ;
}

/*
	 Allocate the memory for the structure d_status, initializes the device and d_status.
	 Returns a pointer to d_status
*/
extern "C"  __host__  mtgp64_kernel_status_t *  init_status (int block_num, int device )
{

	mtgp64_kernel_status_t* d_status;
	ccudaSetDevice(device);
	ccudaMalloc((void**)&d_status, sizeof(mtgp64_kernel_status_t) * block_num);	
	make_constant(MTGP64DC_PARAM_TABLE, block_num);
	return d_status ;
}

/*
	Initialize the seeds associated with each block.
*/

extern "C"  __host__  void init_seeds (mtgp64_kernel_status_t * d_status, int  block_num, uint64_t * seeds ) 
 {
   make_kernel_data64(d_status, MTGP64DC_PARAM_TABLE, block_num,seeds);
 }


/*
	Free the structure d_status from the memory.
*/
extern "C"  __host__  void free_status(mtgp64_kernel_status_t *  d_status ) {
	     ccudaFree(d_status);
 
}

/*
  Return a suitable value for block_num
 */
extern "C"  __host__  int get_suitable_block_num(int device,
					 int *max_block_num,
					 int *mp_num,
					 int word_size,
					 int thread_num,
					 int large_size)
{
  //    DENTER("get_suitable_block");
    hipDeviceProp_t dev;
    hipDevice_t cuDevice;
    int max_thread_dev;
    int max_block, max_block_mem, max_block_dev;
    int major, minor, ver;
    //int regs, max_block_regs;

    ccudaGetDeviceProperties(&dev, device);
    hipDeviceGet(&cuDevice, device);
    hipDeviceComputeCapability(&major, &minor, cuDevice);
    //hipFuncGetAttributes()
#if 0
    if (word_size == 4) {
	regs = 14;
    } else {
	regs = 16;
    }
    max_block_regs = dev.regsPerBlock / (regs * thread_num);
#endif
    max_block_mem = dev.sharedMemPerBlock / (large_size * word_size + 16);
    if (major == 9999 && minor == 9999) {
	return -1;
    }
    ver = major * 100 + minor;
    if (ver <= 101) {
	max_thread_dev = 768;
    } else if (ver <= 103) {
	max_thread_dev = 1024;
    } else if (ver <= 200) {
	max_thread_dev = 1536;
    } else {
	max_thread_dev = 1536;
    }
    max_block_dev = max_thread_dev / thread_num;
    if (max_block_mem < max_block_dev) {
	max_block = max_block_mem;
    } else {
	max_block = max_block_dev;
    }
#if 0
    if (max_block_regs < max_block) {
	max_block = max_block_regs;
    }
#endif
    *max_block_num = max_block;
    *mp_num = dev.multiProcessorCount;
    return max_block * dev.multiProcessorCount;
}


/*

  A simple example

 */
__host__ int sample_cuda(int argc, char** argv)
{
    // LARGE_SIZE is a multiple of 16
    int num_data = 10000000;
    int block_num;
    int block_num_max;
    int num_unit;
    int r;
    mtgp64_kernel_status_t* d_status;
    int device = 0;
    int mb, mp;
    uint64_t * seeds  ;

    ccudaSetDevice(device);

    if (argc >= 2) {
	errno = 0;
	block_num = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	if (BLOCK_NUM_MAX < PARAM_NUM_MAX) {
	    block_num_max = BLOCK_NUM_MAX;
	} else {
	    block_num_max = PARAM_NUM_MAX;
	}
	if (block_num < 1 || block_num > block_num_max) {
	    printf("%s block_num should be between 1 and %d\n",
		   argv[0], block_num_max);
	    return 1;
	}
	errno = 0;
	num_data = strtol(argv[2], NULL, 10);
	if (errno) {
	    printf("%s number_of_block number_of_output\n", argv[0]);
	    return 1;
	}
	argc -= 2;
	argv += 2;
    } else {
	printf("%s number_of_block number_of_output\n", argv[0]);
	block_num = get_suitable_block_num(device,
					   &mb,
					   &mp,
					   sizeof(uint64_t),
					   THREAD_NUM,
					   LARGE_SIZE);
	if (block_num <= 0) {
	    printf("can't calculate suitable number of blocks.\n");
	    return 1;
	}
	printf("the suitable number of blocks for device 0 will be multiple of %d, or %d\n", block_num,(mb - 1) * mp);
	return 1;
    }
    num_unit = LARGE_SIZE * block_num;
    seeds = (uint64_t  * ) malloc( sizeof(uint64_t ) * block_num ) ;
    for (int i=0 ; i< block_num ; i++) seeds[i] = i + 1 ;
    ccudaMalloc((void**)&d_status, sizeof(mtgp64_kernel_status_t) * block_num);
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    printf("number of blocks : %d , number of unit: %d , number of random values: %d\n",block_num,num_unit,num_data);
    make_constant(MTGP64DC_PARAM_TABLE, block_num);
    make_kernel_data64(d_status, MTGP64DC_PARAM_TABLE, block_num,seeds);
    printf("generating 64-bit unsigned random numbers.\n");
    make_uint64_random(d_status, num_data, block_num,1);
    printf("generating double precision floating point random numbers.\n");
    make_double_random(d_status, num_data, block_num,1);

    //finalize
    ccudaFree(d_status);
    free(seeds) ;
    return 0;
}
